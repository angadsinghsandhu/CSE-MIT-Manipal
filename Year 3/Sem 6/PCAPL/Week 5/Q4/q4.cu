#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__host__ __device__ void printMatrix(const char * string, int * A, int width) {
  printf("%s\n", string);
  for (int i = 0; i < width; i++) {
    for (int j = 0; j < width; j++)
      printf("%d, ", A[i * width + j]);
    printf("\n");
  }
  printf("\n");
}
__host__ void clearMatrix(int * A, int width) {
  for (int i = 0; i < width; i++)
    for (int j = 0; j < width; j++)
      A[i * width + j] = 0;
}
__global__ void multiplyMatrixKernel_2a(int * A, int * B, int * C, int width) {
  int row = threadIdx.y;
  int k = 0;
  for (int i = 0; i < width; i++) {
    k = 0;
    for (int j = 0; j < width; j++)
      k += A[row * width + j] * B[i + width * j];
    C[row * width + i] = k;
  }
}
__global__ void multiplyMatrixKernel_2b(int * A, int * B, int * C, int width) {
  int col = threadIdx.x;
  int k = 0;
  for (int i = 0; i < width; i++) {
    k = 0;
    for (int j = 0; j < width; j++)
      k += A[i * width + j] * B[col + j * width];
    C[i * width + col] = k;
  }
}

__global__ void multiplyMatrixKernel_2c(int * A, int * B, int * C, int width) {
  int col = threadIdx.x;
  int row = threadIdx.y;
  int k = 0;
  for (int i = 0; i < width; i++)
    k += A[row * width + i] * B[col + i * width];
  C[row * width + col] = k;
}

void multiplyMatrix(int * h_A, int * h_B, int * h_C, int width) {
  int * d_A, * d_B, * d_C;
  int size = width * width * sizeof(int);
  
  hipMalloc((void ** ) & d_A, size);
  hipMalloc((void ** ) & d_B, size);
  hipMalloc((void ** ) & d_C, size);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);
  
  dim3 dimBlock(1, 1, 1);
  dim3 dimGrid(1, 1, 1);
  
  dimBlock.x = 1;
  dimBlock.y = width;
  dimBlock.z = 1;
  multiplyMatrixKernel_2a <<< dimGrid, dimBlock >>> (d_A, d_B, d_C, width);
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
  printMatrix("A*B: (from 2a kernel): ", h_C, width);
  clearMatrix(h_C, width);
  hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);

  dimBlock.x = width;
  dimBlock.y = 1;
  dimBlock.z = 1;
  multiplyMatrixKernel_2b <<< dimGrid, dimBlock >>> (d_A, d_B, d_C, width);
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
  printMatrix("A*B: (from 2b kernel): ", h_C, width);
  clearMatrix(h_C, width);
  hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);

  dimBlock.x = width;
  dimBlock.y = width;
  dimBlock.z = 1;
  multiplyMatrixKernel_2c <<< dimGrid, dimBlock >>> (d_A, d_B, d_C, width);
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
  printMatrix("A*B: (from 2c kernel): ", h_C, width);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

int main() {
  int * A, * B, * C, k=1, width = 3, size = width * width * sizeof(int);
  A = (int * ) calloc(width * width, sizeof(int));
  B = (int * ) calloc(width * width, sizeof(int));
  C = (int * ) calloc(width * width, sizeof(int));
  
  for (int i = 0; i < width; i++) {
    for (int j = 0; j < width; j++) {
      A[i * width + j] = rand() % 10;
      B[i * width + j] = rand() % 11;
      k++;
    }
  }
  printMatrix("A:", A, width);
  printMatrix("B:", B, width);
  multiplyMatrix(A, B, C, width);
  return 0;
}