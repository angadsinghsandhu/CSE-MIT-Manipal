#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdlib.h>

#include <stdio.h>

__global__
void oddEven(float * arr, int n) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx % 2 == 1 && idx + 1 < n) {
    if (arr[idx] > arr[idx + 1]) {
      float temp = arr[idx];
      arr[idx] = arr[idx + 1];
      arr[idx + 1] = temp;
    }
  }
}
__global__
void evenOdd(float * arr, int n) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx % 2 == 0 && idx + 1 < n) {
    if (arr[idx] > arr[idx + 1]) {
      float temp = arr[idx];
      arr[idx] = arr[idx + 1];
      arr[idx + 1] = temp;
    }
  }
}
void oddEvenTranspositionSort(float * arr, int n) {
  int size = n * sizeof(float);
  float * d_arr;
  hipMalloc((void ** ) & d_arr, size);
  hipMemcpy(d_arr, arr, size, hipMemcpyHostToDevice);
  for (int i = 0; i <= n / 2; i++) {
    oddEven << < 1, n >>> (d_arr, n);
    evenOdd << < 1, n >>> (d_arr, n);
  }
  hipMemcpy(arr, d_arr, size, hipMemcpyDeviceToHost);
  hipFree(d_arr);
}
int main() {
  float * h_arr;
  int n = 5;
  int size = n * sizeof(float);
  h_arr = (float * ) malloc(size);
  for (int i = 0; i < 5; i++) {
    h_arr[i] = rand() % 50;
  }
  printf("unsorted_arr: ");
  for (int i = 0; i < n; i++) {
    printf("%f, ", h_arr[i]);
  }
  printf("\n\n");
  oddEvenTranspositionSort(h_arr, n);
  printf("sorted_arr: ");
  for (int i = 0; i < n; i++) {
    printf("%f, ", h_arr[i]);
  }
  printf("\n");
  return 0;
}