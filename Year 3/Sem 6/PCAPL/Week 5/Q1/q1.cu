#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

 // Q1a
__global__ void vecAddKernel_1a(float * A, float * B, float * C) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  C[idx] = A[idx] + B[idx];
}

 // Q1b
__global__ void vecAddKernel_1b(float * A, float * B, float * C) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  C[idx] = A[idx] + B[idx];
}

 // Q1c
__global__
void vecAddKernel_1c(float * A, float * B, float * C, int n) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < n) C[idx] = A[idx] + B[idx];
}

void vecAdd(float * A, float * B, float * C, int n) {
  int size = n * sizeof(float);
  float *d_A, *d_B, *d_C;
  hipMalloc((void ** ) & d_A, size);
  hipMalloc((void ** ) & d_B, size);
  hipMalloc((void ** ) & d_C, size);
  
  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

  printf("A: ");
  for (int i = 0; i < n; i++)
    printf("%f, ", A[i]);
  printf("\n");

  printf("B: ");
  for (int i = 0; i < n; i++)
    printf("%f, ", B[i]);
  printf("\n\n");

  vecAddKernel_1a <<< n, 1 >>> (d_A, d_B, d_C);
  hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
  printf("A + B (from 1a kernel): ");
  for (int i = 0; i < n; i++)
    printf("%f, ", C[i]);
  printf("\n");

  vecAddKernel_1b <<< 1, n >>> (d_A, d_B, d_C);
  hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
  printf("A+B (from 1b kernel): ");
  for (int i = 0; i < n; i++)
    printf("%f, ", C[i]);
  printf("\n");

  vecAddKernel_1c << < ceil(n / 256.0), n >>> (d_A, d_B, d_C, n);
  hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
  printf("A+B (from 1c kernel): ");

  for (int i = 0; i < n; i++)
    printf("%f, ", C[i]);
  printf("\n");

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

int main() {
  float * h_A, * h_B, * h_C;
  int n = 5, size = n * sizeof(float);
  h_A = (float * ) malloc(size);
  h_B = (float * ) malloc(size);
  h_C = (float * ) malloc(size);

  for (int i = 0; i < n; i++) {
    h_A[i] = (i + 1) * 10;
    h_B[i] = i + 1;
  }
  vecAdd(h_A, h_B, h_C, n);
  return 0;
}