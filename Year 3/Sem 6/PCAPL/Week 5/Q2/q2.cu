#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void selectionSortKernel<<< 1, n >>>(float * arr, float * lst, int n) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  float key = arr[idx];
  int pos = 0;
  for (int i = 0; i < n; i++) {
    if (arr[i] < key || (arr[i] == key && i < idx))
      pos++;
  }
  lst[pos] = key;
}

void selectionSort(float * arr, float * lst, int n) {
  int size = n * sizeof(float);
  float *d_arr, *d_lst;

  hipMalloc((void ** ) & d_arr, size);
  hipMalloc((void ** ) & d_lst, size);
  hipMemcpy(d_arr, arr, size, hipMemcpyHostToDevice);
  selectionSortKernel(d_arr, d_lst, n);
  hipMemcpy(lst, d_lst, size, hipMemcpyDeviceToHost);
  hipFree(d_arr);
  hipFree(d_lst);
}

int main() {
  float * h_arr, * h_lst;
  int n = 5, size = n * sizeof(float);
  h_arr = (float * ) malloc(size);
  h_lst = (float * ) malloc(size);

  for (int i = 0; i < 5; i++)
    h_arr[i] = rand() % 50;
  
  selectionSort(h_arr, h_lst, n);
  printf("arr: ");
  for (int i = 0; i < n; i++)
    printf("%f, ", h_arr[i]);
  printf("\n\n");

  printf("lst: ");
  for (int i = 0; i < n; i++)
    printf("%f, ", h_lst[i]);
  printf("\n");
  return 0;
}