#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void selectionSortKernel<<< 1, n >>>(float * unsorted_arr, float * sorted_arr, int n) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  float key = unsorted_arr[idx];
  int pos = 0;
  for (int i = 0; i < n; i++) {
    if (unsorted_arr[i] < key || (unsorted_arr[i] == key && i < idx))
      pos++;
  }
  sorted_arr[pos] = key;
}

void selectionSort(float * unsorted_arr, float * sorted_arr, int n) {
  int size = n * sizeof(float);
  float * d_unsorted_arr;
  float * d_sorted_arr;

  hipMalloc((void ** ) & d_unsorted_arr, size);
  hipMalloc((void ** ) & d_sorted_arr, size);
  hipMemcpy(d_unsorted_arr, unsorted_arr, size, hipMemcpyHostToDevice);
  selectionSortKernel(d_unsorted_arr, d_sorted_arr, n);
  hipMemcpy(sorted_arr, d_sorted_arr, size, hipMemcpyDeviceToHost);
  hipFree(d_unsorted_arr);
  hipFree(d_sorted_arr);
}

int main() {
  float * h_unsorted_arr, * h_sorted_arr;
  int n = 5;
  int size = n * sizeof(float);
  h_unsorted_arr = (float * ) malloc(size);
  h_sorted_arr = (float * ) malloc(size);

  for (int i = 0; i < 5; i++)
    h_unsorted_arr[i] = rand() % 50;
  
  selectionSort(h_unsorted_arr, h_sorted_arr, n);
  printf("unsorted_arr: ");
  for (int i = 0; i < n; i++)
    printf("%f, ", h_unsorted_arr[i]);
  printf("\n\n");

  printf("sorted_arr: ");
  for (int i = 0; i < n; i++)
    printf("%f, ", h_sorted_arr[i]);
  printf("\n");
  return 0;
}