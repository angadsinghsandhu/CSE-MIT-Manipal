#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>



/*
Sparce Matrix Multiplication
*/

__global__ void spvm(int num_rows, int *data, int *col_indices, int *row_ptr, int *x, int *y)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < num_rows)
    {
        int dot=0;
        int row_start = row_ptr[row];
        int row_end = row_ptr[row+1];

        for(int elem= row_start; elem<row_end; elem++)
        {
            dot += data[elem] * x[col_indices[elem]];
        }
        y[row] = dot;
    }
}

int main()
{
    int m,n;
    printf("Enter rows and col : ");
    scanf("%d%d",&m,&n);

    int** arr = (int**)malloc(m * sizeof(int*));
    for (int i = 0; i < m; i++)
        arr[i] = (int*)malloc(n * sizeof(int));

    printf("Enter %d elements : \n",m*n);
    for(int i=0;i<m;i++)
        for(int j=0;j<n;j++)
            scanf("%d",&arr[i][j]);

    int *x = (int*)malloc(n*sizeof(int));
    int *y = (int*)malloc(n*sizeof(int));
    printf("Enter vector X(%d elements) : ",n);
    for(int i=0;i<n;i++)
        scanf("%d",&x[i]);

    int data[20], col_indices[20], row_ptr[20];
    int count = 0;
    row_ptr[0] = 0;
    for(int i=0;i<m;i++)
    {
        int c = 0;
        for(int j=0;j<n;j++)
            if(arr[i][j] != 0)
            {
                c++;
                data[count] = arr[i][j];
                col_indices[count++] = j;
            }

        row_ptr[i+1] = row_ptr[i] + c;  
    }

    int *d_data, *d_col_indices, *d_row_ptr, *d_x, *d_y;
    hipMalloc((void**)&d_data,count*sizeof(int));
    hipMalloc((void**)&d_col_indices,count*sizeof(int));
    hipMalloc((void**)&d_row_ptr,(m+1)*sizeof(int));
    hipMalloc((void**)&d_x,n*sizeof(int));
    hipMalloc((void**)&d_y,m*sizeof(int));

    hipMemcpy(d_data,data,count*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_col_indices,col_indices,count*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_row_ptr,row_ptr,(m+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_x,x,n*sizeof(int),hipMemcpyHostToDevice);
    
    spvm<<<1,m>>>(m,d_data,d_col_indices,d_row_ptr,d_x,d_y);
    
    hipMemcpy(y,d_y,m*sizeof(int),hipMemcpyDeviceToHost);
    
    printf("\n\n");
    for(int i=0;i<m;i++)
        printf("%d  ",y[i]);
    printf("\n\n");

    hipFree(d_data);
    hipFree(d_col_indices);
    hipFree(d_row_ptr);
    hipFree(d_x);
}

