
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

#define MASK_WIDTH 5
#define WIDTH 10

/* 
do 1D convolution
1 - with mask in global memory
2 - with mask in shared memory
3 - with mask in constant memory
*/

__constant__ int d_Mc[MASK_WIDTH];

__global__ void convolution_global_memory(int *A, int *M, int *R){
	
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	float val = 0;
	int start = i-(MASK_WIDTH/2);
    

	for(int j =0; j<MASK_WIDTH;j++){
		if(start+j >=0 && start+j < WIDTH){
			val+= A[start+j]*M[j];
		}
	}
	R[i]=val;
}

__global__ void convolution_shared_memory(int *A, int *M, int *R){
    __shared__ int d_Ms[MASK_WIDTH];
    for(int i=0; i<MASK_WIDTH; i++)
        d_Ms[i] = M[i];
    
    int i = blockIdx.x*blockDim.x+threadIdx.x;
	float val = 0;
	int start = i-(MASK_WIDTH/2);
    

	for(int j =0; j<MASK_WIDTH;j++){
		if(start+j >=0 && start+j < WIDTH){
			val+= A[start+j]*d_Ms[j];
		}
	}
	R[i]=val;
}

__global__ void convolution_constant_memory(int *A, int *R){
    int i = blockIdx.x*blockDim.x+threadIdx.x;
	float val = 0;
	int start = i-(MASK_WIDTH/2);
    
	for(int j =0; j<MASK_WIDTH;j++){
        if(start+j >=0 && start+j < WIDTH){
            val+= A[start+j] * d_Mc[j];
		}
	}
	R[i]=val;
}

int main(){
    int A[WIDTH], M[MASK_WIDTH], R[WIDTH];
    int *d_A, *d_M, *d_R;

    for(int i=0; i<WIDTH; i++){
        A[i] = rand() % 10;
        R[i] = 0;

        if(i < MASK_WIDTH)
            M[i] = rand() % 10;
    }

    printf("Input Matrix : ");
    for(int i=0; i<WIDTH; i++){
        printf("%d ", A[i]);
    }

    printf("\nMask Matrix : ");
    for(int i=0; i<MASK_WIDTH; i++){
        printf("%d ", M[i]);
    }
    printf("\n\n");

    hipError_t err;

    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 1 : %s\n", hipGetErrorString(err));

    hipMalloc((void**)&d_A, WIDTH*sizeof(int));
    hipMalloc((void**)&d_M, MASK_WIDTH*sizeof(int));
    hipMalloc((void**)&d_R, WIDTH*sizeof(int));

    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 2 : %s\n", hipGetErrorString(err));

    hipMemcpy(d_A, A, WIDTH*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_M, M, MASK_WIDTH*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_R, R, WIDTH*sizeof(int), hipMemcpyHostToDevice);

    if(hipGetLastError() != hipSuccess)
        printf("CUDA Error Occured 3 : %s\n", hipGetErrorString(hipGetLastError()));


    // Q2a
    convolution_global_memory<<<1, WIDTH>>>(d_A, d_M, d_R);

    // // Q2b
    // convolution_shared_memory<<<1, WIDTH>>>(d_A, d_M, d_R);

    // // Q2c
    // cudaMemcpyToSymbol(d_Mc, M, MASK_WIDTH*sizeof(int));
    // convolution_constant_memory<<<1, WIDTH>>>(d_A, d_R);

    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 4 : %s\n", hipGetErrorString(err));

    hipMemcpy(R, d_R, WIDTH*sizeof(int), hipMemcpyDeviceToHost);

    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 5 : %s\n", hipGetErrorString(err));


    printf("Resultant Matrix after convolution : ");
    for(int i=0; i<WIDTH; i++){
        printf("%d ", R[i]);
    }
    printf("\n");

    hipFree(d_A);
    hipFree(d_M);
    hipFree(d_R);
    return 0;
}
