#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define N 1024

__global__ void CUDACount(char* A, int* cnt){
    int idx = threadIdx.x;

    if(A[idx] == 'a')
        atomicAdd(cnt, 1);
}

int main(){
    char A[N];
    char *d_A;

    int count=0, *d_count, res;

    hipError_t err = hipGetLastError();

    printf("Enter String : ");
    scanf("%s", A);
    printf("String : %s\n", A);

    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 1 : %s\n", hipGetErrorString(err));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 2 : %s\n", hipGetErrorString(err));

    hipMalloc((void**)&d_A, strlen(A)*sizeof(char));
    hipMalloc((void**)&d_count, sizeof(int));
    hipMalloc((void**)&res, sizeof(int));

    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 3 : %s\n", hipGetErrorString(err));

    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);
    
    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 3.5 : %s\n", hipGetErrorString(err));
        
    hipMemcpy(d_A, A, strlen(A)*sizeof(char), hipMemcpyHostToDevice);


    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 4 : %s\n", hipGetErrorString(err));

    CUDACount<<<1, strlen(A)>>>(d_A, d_count);

    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 5 : %s\n", hipGetErrorString(err));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    

    hipMemcpy(&res, d_count, sizeof(int), hipMemcpyDeviceToHost);
    printf("Total Occurances of 'a' = %d\n", res);

    hipFree(d_A);
    hipFree(d_count);
    return 0;

}
