#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define N 1024

__global__ void CUDACount(char* A, char* B, int* len, int* wordLen, int* cnt){
    int idx = threadIdx.x, flag=1;
    
    if(idx + *wordLen <= *len){
        for(int i=0; i<*wordLen; i++){
            if(A[idx + i] != B[i]){
                flag = 0;
                break;
            }
        }

        if(flag==1)
            atomicAdd(cnt, 1);
    }
}

int main(){
    char A[N], B[N];
    char *d_A, *d_B;

    int count=0, len, wordLen,res;
    int *d_count, *d_len, *d_wordLen;

    hipError_t err = hipGetLastError();

    printf("Enter String : ");
    scanf("%s", A);
    printf("String : %s\n\n", A);

    printf("Enter Word to be searched in String : ");
    scanf("%s", B);
    printf("Word : %s\n\n", B);

    len = strlen(A);
    wordLen = strlen(B);

    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 1 : %s\n", hipGetErrorString(err));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 2 : %s\n", hipGetErrorString(err));

    hipMalloc((void**)&d_A, strlen(A)*sizeof(char));
    hipMalloc((void**)&d_B, strlen(B)*sizeof(char));
    hipMalloc((void**)&d_count, sizeof(int));
    hipMalloc((void**)&d_len, sizeof(int));
    hipMalloc((void**)&d_wordLen, sizeof(int));
    hipMalloc((void**)&res, sizeof(int));

    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 3 : %s\n", hipGetErrorString(err));

    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_len, &len, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_wordLen, &wordLen, sizeof(int), hipMemcpyHostToDevice);
    
    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 3.5 : %s\n", hipGetErrorString(err));
        
    hipMemcpy(d_A, A, strlen(A)*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, strlen(B)*sizeof(char), hipMemcpyHostToDevice);


    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 4 : %s\n", hipGetErrorString(err));

    CUDACount<<<1, strlen(A)>>>(d_A, d_B, d_len, d_wordLen, d_count);

    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 5 : %s\n", hipGetErrorString(err));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    

    hipMemcpy(&res, d_count, sizeof(int), hipMemcpyDeviceToHost);
    printf("Total Occurances of '%s' = %d\n", B, res);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_len);
    hipFree(d_wordLen);
    hipFree(d_count);
    return 0;

}
